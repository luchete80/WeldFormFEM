#include "hip/hip_runtime.h"
#include "NodeData.cuh"


 void AllocateNodeData(NodeData *node, const int &nc){
  //node_count = nc;
  hipMalloc((void **)&node->x, nc * sizeof (double3));
  hipMalloc((void **)&node->v, nc * sizeof (double3));
  hipMalloc((void **)&node->a, nc * sizeof (double3));
  hipMalloc((void **)&node->u, nc * sizeof (double3));
 }