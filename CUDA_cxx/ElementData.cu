#include "ElementData.cuh"


void AllocateElements(ElementData *elem, const int &dim, const int &el_count){
  
  hipMalloc((void **)&elem->pressure, el_count * sizeof (double)); //8 values per dim 
  
  //Assuming reduced integration
  //hipMalloc((void **)&elem->dHxy_detJ, el_count * dim * nodxelem* sizeof (double)); //8 values per dim 
}