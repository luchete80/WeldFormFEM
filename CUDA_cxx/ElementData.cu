#include "ElementData.cuh"


//// FOR A SINGLE ELEMENT DATA

void AllocateElementData(ElementData *elem, const int &dim, const int &el_count, const int &gp, const int &nodxelem){
  
  hipMalloc((void **)&elem->pressure, el_count * sizeof (double)); //8 values per dim 
  hipMalloc((void **)&elem->gausspc, el_count * sizeof (int)); 
  
  hipMalloc((void **)&elem->dHxy_detJ, el_count*gp*dim*nodxelem*sizeof (double));

  //ATTENTION; THIS SHOULD BE CHANGE WHEN ADDING DIFFERENT TYPEOFELEMENTS
  hipMalloc((void **)&elem->elnod, el_count * nodxelem * sizeof (unsigned long));
  hipMalloc((void **)&elem->elnod_offset, el_count * sizeof (unsigned long));
  
  
  //Assuming reduced integration
  //hipMalloc((void **)&elem->dHxy_detJ, el_count * dim * nodxelem* sizeof (double)); //8 values per dim 


 
     // hipMalloc((void **)&elem->elnod(el_count,nodxelem))

    // hipMalloc((void **)&elem->dof(el_count,dim*nodxelem))
    // hipMalloc((void **)&elem->vol(el_count))
    // hipMalloc((void **)&elem->vol_inc(el_count))
    // hipMalloc((void **)&elem->vol_0(el_count))
    // hipMalloc((void **)&elem->x2(el_count,nodxelem,dim))
    // hipMalloc((void **)&elem->jacob(el_count,gp,dim,dim))
    // hipMalloc((void **)&elem->detj(el_count,gp))
    // hipMalloc((void **)&elem->sigma_eq(el_count,gp)) !But is constant??
    // hipMalloc((void **)&elem->dHxy(el_count,gp,dim,nodxelem))
    // hipMalloc((void **)&elem->dHxy_detJ(el_count,gp,dim,nodxelem)) !!!! STORE LIKE THIS TO SAVE CALCULATION TIME (THIS IS USED  TO CALC FORCES INTEGRATING IT )
    // hipMalloc((void **)&elem->dHxy0(el_count,gp,dim,nodxelem)) !!!USED FOR DEFORMATION GRADIENT ONLY FOR FULL INTEGRATION ELEMENTS 
    // hipMalloc((void **)&elem->dHrs(el_count,gp,dim,nodxelem))
    // hipMalloc((void **)&elem->sigma(el_count,gp,dim,dim))  !!!THIS IS A DIMxDIM SYMMETRIC TENSOR

    // hipMalloc((void **)&elem->uele (el_count,dim*nodxelem,1)) 

    // hipMalloc((void **)&elem->vele (el_count,dim*nodxelem,1)) 
    
    // hipMalloc((void **)&elem->mass(el_count)) !Mass matrix    
    
    // hipMalloc((void **)&elem->c_s(el_count,gp))
    // hipMalloc((void **)&elem->p_visc(el_count,gp))
    // hipMalloc((void **)&elem->e_length(el_count))

    // hipMalloc((void **)&elem->matm(el_count,nodxelem,nodxelem)) !Mass matrix
    // hipMalloc((void **)&elem->math(el_count,gp,1,nodxelem)) !Mass matrix
    
    // hipMalloc((void **)&elem->hourg_nodf(el_count,nodxelem,dim)) !AS 1 COLUMN OR NOT????? Mass matrix
    
    // hipMalloc((void **)&elem->f_int(el_count,nodxelem,dim))
    // hipMalloc((void **)&elem->f_ext(el_count,nodxelem,dim))
    
    // hipMalloc((void **)&elem->rho(el_count,gp)) !AT FIRST ONLY ONE POINT
    // hipMalloc((void **)&elem->rho_0(el_count,gp))
    // hipMalloc((void **)&elem->pressure(el_count,gp))
    // hipMalloc((void **)&elem->cs(el_count))
    // hipMalloc((void **)&elem->shear_stress(el_count,gp, dim,dim))
    // hipMalloc((void **)&elem->str_rate(el_count,gp, dim,dim))
    // hipMalloc((void **)&elem->str_inc(el_count,gp, dim,dim))
    // hipMalloc((void **)&elem->rot_rate(el_count,gp, dim,dim))
      
    // if (Dim .eq. 2) then
      // hipMalloc((void **)&elem->bl (el_count,gp,3,dim*nodxelem))
      // hipMalloc((void **)&elem->bnl(el_count,gp, 4,dim*nodxelem))
      // hipMalloc((void **)&elem->strain(el_count,gp, 4,1))
      // !hipMalloc((void **)&elem->str_rate(el_count,gp, 4,1))
      // !hipMalloc((void **)&elem->rot_rate(el_count,gp, 4,1))
    // else 
      // hipMalloc((void **)&elem->bl (el_count,gp,6,dim*nodxelem)) 
      // hipMalloc((void **)&elem->strain(el_count,gp, 6,1)) !!VECTORIZED 
      // !hipMalloc((void **)&elem->str_rate(el_count,gp, 6,1))
      // !hipMalloc((void **)&elem->rot_rate(el_count,gp, 6,1))
    // end if 
    
    // elem->gausspc(:) = gp
    
}